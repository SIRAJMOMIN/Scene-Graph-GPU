#include "hip/hip_runtime.h"

/*
	CS 6023 Assignment 3.
	Do not make any changes to the boiler plate code or the other files in the folder.
	Use hipFree to deallocate any memory not in usage.
	Optimize as much as possible.
*/

#include "SceneNode.h"
#include <queue>
#include "Renderer.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <chrono>


__global__ void dkernel1(int limit,int *gstart,int *gt,int *gcount,int *gchild,int *gglobalx,int *gglobaly)
{
    unsigned id =blockIdx.x*blockDim.x+threadIdx.x;
    if(id<limit){
    int ver=gt[id*3];
    int di=gt[id*3+1];
    int n=gt[id*3+2];
    int childcount=gcount[ver];
    int s=gstart[ver];
    if(di==2)
    { 
     atomicSub(&gglobaly[ver],n);
    }
    else if(di==3)
    { 
      atomicAdd(&gglobaly[ver],n);
    }
    else if(di==1) //DOWN
    {
      atomicAdd(&gglobalx[ver],n);
    }
    else if(di==0) 
    {
       atomicSub(&gglobalx[ver],n);
    }
    for(int i=s;i<s+childcount;i++)
    {
        int f=gchild[i];
        if(di==2)
        { 
           atomicSub(&gglobaly[f],n);
        }
        else if(di==3)
        {
          atomicAdd(&gglobaly[f],n);
        }
        else if(di==1) 
        {
          atomicAdd(&gglobalx[f],n);
        }
        else if(di==0) 
        {
          atomicSub(&gglobalx[f],n);
        }
    }
    }
}

__global__ void dkernel2(int **gm,int *meshx,int *meshy,int *gx,int *gy,int *gopacity,int frameSizeX,int frameSizeY,int V,int *final)
{
    unsigned id=blockIdx.x*blockDim.x+threadIdx.x;
    if(id<(frameSizeX*frameSizeY))
    {
        int row=id/frameSizeY;
        int col=id%frameSizeY;
        int meshopacity=-1;
        bool x=false;
        for(int k=0;k<V;k++)
        {
            if(row<meshx[k]+gx[k] && row>=gx[k])
            {
                if(col<meshy[k]+gy[k] && col>=gy[k])
                {
                  x=meshopacity<gopacity[k];
                  if(x)
                  {
                      meshopacity=gopacity[k];
                      int mx=row-gx[k];
                      int my=col-gy[k];
                      final[row*frameSizeY+col]=gm[k][mx*meshy[k]+my];

                  }
                }
            }
        }
    }

}




void readFile (const char *fileName, std::vector<SceneNode*> &scenes, std::vector<std::vector<int> > &edges, std::vector<std::vector<int> > &translations, int &frameSizeX, int &frameSizeY) {
	/* Function for parsing input file*/

	FILE *inputFile = NULL;
	// Read the file for input.
	if ((inputFile = fopen (fileName, "r")) == NULL) {
		printf ("Failed at opening the file %s\n", fileName) ;
		return ;
	}

	// Input the header information.
	int numMeshes ;
	fscanf (inputFile, "%d", &numMeshes) ;
	fscanf (inputFile, "%d %d", &frameSizeX, &frameSizeY) ;


	// Input all meshes and store them inside a vector.
	int meshX, meshY ;
	int globalPositionX, globalPositionY; // top left corner of the matrix.
	int opacity ;
	int* currMesh ;
	for (int i=0; i<numMeshes; i++) {
		fscanf (inputFile, "%d %d", &meshX, &meshY) ;
		fscanf (inputFile, "%d %d", &globalPositionX, &globalPositionY) ;
		fscanf (inputFile, "%d", &opacity) ;
		currMesh = (int*) malloc (sizeof (int) * meshX * meshY) ;
		for (int j=0; j<meshX; j++) {
			for (int k=0; k<meshY; k++) {
				fscanf (inputFile, "%d", &currMesh[j*meshY+k]) ;
			}
		}
		//Create a Scene out of the mesh.
		SceneNode* scene = new SceneNode (i, currMesh, meshX, meshY, globalPositionX, globalPositionY, opacity) ;
		scenes.push_back (scene) ;
	}

	// Input all relations and store them in edges.
	int relations;
	fscanf (inputFile, "%d", &relations) ;
	int u, v ;
	for (int i=0; i<relations; i++) {
		fscanf (inputFile, "%d %d", &u, &v) ;
		edges.push_back ({u,v}) ;
	}

	// Input all translations.
	int numTranslations ;
	fscanf (inputFile, "%d", &numTranslations) ;
	std::vector<int> command (3, 0) ;
	for (int i=0; i<numTranslations; i++) {
		fscanf (inputFile, "%d %d %d", &command[0], &command[1], &command[2]) ;
		translations.push_back (command) ;
	}
}


void writeFile (const char* outputFileName, int *hFinalPng, int frameSizeX, int frameSizeY) {
	/* Function for writing the final png into a file.*/
	FILE *outputFile = NULL;
	if ((outputFile = fopen (outputFileName, "w")) == NULL) {
		printf ("Failed while opening output file\n") ;
	}

	for (int i=0; i<frameSizeX; i++) {
		for (int j=0; j<frameSizeY; j++) {
			fprintf (outputFile, "%d ", hFinalPng[i*frameSizeY+j]) ;
		}
		fprintf (outputFile, "\n") ;
	}
}


int main (int argc, char **argv) {

	// Read the scenes into memory from File.
	const char *inputFileName = argv[1] ;
	int* hFinalPng ;

	int frameSizeX, frameSizeY ;
	std::vector<SceneNode*> scenes ;
	std::vector<std::vector<int> > edges ;
	std::vector<std::vector<int> > translations ;
	readFile (inputFileName, scenes, edges, translations, frameSizeX, frameSizeY) ;
	hFinalPng = (int*) malloc (sizeof (int) * frameSizeX * frameSizeY) ;

	// Make the scene graph from the matrices.
    Renderer* scene = new Renderer(scenes, edges) ;

	// Basic information.
	int V = scenes.size () ;
	int E = edges.size () ;
	int numTranslations = translations.size () ;

	// Convert the scene graph into a csr.
	scene->make_csr () ; // Returns the Compressed Sparse Row representation for the graph.
	int *hOffset = scene->get_h_offset () ;
	int *hCsr = scene->get_h_csr () ;
	int *hOpacity = scene->get_opacity () ; // hOpacity[vertexNumber] contains opacity of vertex vertexNumber.
	int **hMesh = scene->get_mesh_csr () ; // hMesh[vertexNumber] contains the mesh attached to vertex vertexNumber.
	int *hGlobalCoordinatesX = scene->getGlobalCoordinatesX () ; // hGlobalCoordinatesX[vertexNumber] contains the X coordinate of the vertex vertexNumber.
	int *hGlobalCoordinatesY = scene->getGlobalCoordinatesY () ; // hGlobalCoordinatesY[vertexNumber] contains the Y coordinate of the vertex vertexNumber.
	int *hFrameSizeX = scene->getFrameSizeX () ; // hFrameSizeX[vertexNumber] contains the vertical size of the mesh attached to vertex vertexNumber.
	int *hFrameSizeY = scene->getFrameSizeY () ; // hFrameSizeY[vertexNumber] contains the horizontal size of the mesh attached to vertex vertexNumber.

	auto start = std::chrono::high_resolution_clock::now () ;


	// Code begins here.
	// Do not change anything above this comment.
  std::vector<int> child;
  std::vector<int> count;
  std::vector<int> nstart;
  nstart.push_back(0);
  int x=0;

  for (int i=0;i<V;i++)
  {
    int ncount=0;
    std::queue<int>q;
    q.push(i);
    while(!q.empty())
    {
      int x=q.front();
      q.pop();
      int start=hOffset[x];
      int diff=hOffset[x+1]-start;
      for(int j=start;j<start+diff;j++)
      {
        q.push(hCsr[j]);
        child.push_back(hCsr[j]);
        ncount++;
      }
    }
    x+=ncount;
    count.push_back(ncount);
    nstart.push_back(x);
  }
  //dkernel-1
  int *cchild=(int*)malloc(sizeof(int)*child.size());
  for(int i=0;i<child.size();i++)
  cchild[i]=child[i];
  int blocks=ceil(numTranslations/1024.0);
  int *gchild;
  hipMalloc(&gchild,child.size()*sizeof(int));
  hipMemcpy(gchild,cchild,child.size()*sizeof(int),hipMemcpyHostToDevice);
  int *gglobalx;
  int *gglobaly;
  hipMalloc(&gglobalx,V*sizeof(int));
  hipMalloc(&gglobaly,V*sizeof(int));
  hipMemcpy(gglobalx,hGlobalCoordinatesX,V*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(gglobaly,hGlobalCoordinatesY,V*sizeof(int),hipMemcpyHostToDevice);
  int *gcount;
  int *ccount=(int*)malloc(sizeof(int)*count.size());
  for(int i=0;i<count.size();i++)
  ccount[i]=count[i];
  hipMalloc(&gcount,count.size()*sizeof(int));
  hipMemcpy(gcount,ccount,count.size()*sizeof(int),hipMemcpyHostToDevice);
  int *ct=(int*)malloc(numTranslations*3*sizeof(int));
  int k=0;
  for(int i=0;i<translations.size();i++)
  {
      for(int j=0;j<3;j++)
      ct[k++]=translations[i][j];
  }
  int *gt;
  hipMalloc(&gt,sizeof(int)*numTranslations*3);
  hipMemcpy(gt,ct,sizeof(int)*numTranslations*3,hipMemcpyHostToDevice);
  int *gstart;
  int *cstart=(int*)malloc(sizeof(int)*nstart.size());
  for(int i=0;i<nstart.size();i++)
  cstart[i]=nstart[i];
  hipMalloc(&gstart,nstart.size()*sizeof(int));
  hipMemcpy(gstart,cstart,sizeof(int)*nstart.size(),hipMemcpyHostToDevice);
  dkernel1<<<blocks,1024>>>(numTranslations,gstart,gt,gcount,gchild,gglobalx,gglobaly);
  hipMemcpy(hGlobalCoordinatesX,gglobalx,V*sizeof(int),hipMemcpyDeviceToHost);
  hipMemcpy(hGlobalCoordinatesY,gglobaly,V*sizeof(int),hipMemcpyDeviceToHost);
  free(cchild);
  free(cstart);
  free(ct);
  free(ccount);
  hipFree(gchild);
  hipFree(gstart);
  hipFree(gt);
  hipFree(gcount);

  //dkernel-2
  int **gm;
  hipMalloc(&gm,V*sizeof(int*));
  for(int i=0;i<V;i++){
     int *dummy;
     hipMalloc(&dummy,hFrameSizeX[i]*hFrameSizeY[i]*sizeof(int));
     hipMemcpy(dummy,hMesh[i],hFrameSizeX[i]*hFrameSizeY[i]*sizeof(int),hipMemcpyHostToDevice);
     hipMemcpy(&gm[i],&dummy,sizeof(int*),hipMemcpyHostToDevice);}
  int nblocks=ceil(frameSizeX*frameSizeY/1024.0);
  int *meshx;
  int *meshy;
  hipMalloc(&meshx,V*sizeof(int));
  hipMalloc(&meshy,V*sizeof(int));
  hipMemcpy(meshx,hFrameSizeX,V*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(meshy,hFrameSizeY,V*sizeof(int),hipMemcpyHostToDevice);
  int *gx;
  int *gy;
  hipMalloc(&gx,V*sizeof(int));
  hipMalloc(&gy,V*sizeof(int));
  hipMemcpy(gx,hGlobalCoordinatesX,V*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(gy,hGlobalCoordinatesY,V*sizeof(int),hipMemcpyHostToDevice);
  int *gopacity;
  hipMalloc(&gopacity,V*sizeof(int));
  hipMemcpy(gopacity,hOpacity,V*sizeof(int),hipMemcpyHostToDevice);
  int *final;
  hipMalloc(&final,frameSizeX*frameSizeY*sizeof(int));
  dkernel2<<<nblocks,1024>>>(gm,meshx,meshy,gx,gy,gopacity,frameSizeX,frameSizeY,V,final);
  hipMemcpy(hFinalPng,final,frameSizeX*frameSizeY*sizeof(int),hipMemcpyDeviceToHost);
 // Do not change anything below this comment.
 // Code ends here.

	auto end  = std::chrono::high_resolution_clock::now () ;

	std::chrono::duration<double, std::micro> timeTaken = end-start;

	printf ("execution time : %f\n", timeTaken) ;
	// Write output matrix to file.
	const char *outputFileName = argv[2] ;
	writeFile (outputFileName, hFinalPng, frameSizeX, frameSizeY) ;

}
